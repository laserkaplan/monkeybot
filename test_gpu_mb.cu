#include <iostream>

#include "monkeybot.cuh"

#define N 13

void seta(char *a) {
    a[0] = 'H';
    a[1] = 'e';
    a[2] = 'l';
    a[3] = 'l';
    a[4] = 'o';
    a[5] = ' ';
    a[6] = 'W';
    a[7] = 'o';
    a[8] = 'r';
    a[9] = 'l';
    a[10] = 'd';
    a[11] = '!';
    a[12] = '\0';
}

int main(void) {
    // host variables
    char *a, *b;
    
    // device copies
    char *d_a, *d_b;
    
    // define size
    int size = N * sizeof(char);

    // allocate memory on device
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    // initialize host variables
    a = (char *)malloc(size);
    seta(a);
    b = (char *)malloc(size);

    // copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // execute device method
    mb::copychar<<<1,N>>>(d_a, d_b);

    // copy results back to host
    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    // print result
    std::cout << a << std::endl;
    std::cout << b << std::endl;

    // free memory
    free(a);
    free(b);
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
